#include "animation.h"

#include <hip/hip_runtime.h>

animation::animation( std::size_t width, std::size_t height) :
        x( width),
        y( height) {
    hipMallocManaged( & pixels_, size() );
}

animation::~animation() {
    hipFree( pixels_);
}
